/*****************************************************************************
* Dai NAKAI
* Three dimensional Droplets distribution and its hologram generating software
* 2021/5/22
* 
******************************************************************************/

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <cstdlib>
#include <sys/stat.h>
#include <time.h>
#include <hipfft/hipfft.h>
using namespace std;

#define WAVE_LEN 0.6328f
#define PI 3.14159265f
#define DX 10.0f

//CUDA function error chech macro
#define CHECK(call)                                                             \
{                                                                               \
    const hipError_t error = call;                                             \
    if(error != hipSuccess){                                                   \
        printf("Error: %s:%d, ",__FILE__, __LINE__);                            \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));      \
        exit(1);                                                                \
    }                                                                           \
}

//Confirm ref image when you change width or height. 
//ref-img should be half of output.
const char* ref_img_path = "../1024.bmp";
// const char* output_dir = "../holograms/train/close_holo";
char output_dir_num[100];
unsigned char header_buf[1078];
char output_path[100];

const int height = 2048;
const int width = 2048;
const int depth = 1024;
// const int particle_num = 17;
// const int image_count = 3000;

//All units provided by micro meter
const float mean_diam = 50.0;
const float sd_diam = 10.0;
const float dist_to_cam = 50000.0;

const float peak_bright = 127;

void particle_diam_dist (float *diam, int index);
void particle_posi_dist (float *x, float *y, float*z, int particle_num);
void z_axis_sort (float *array, int left, int right);
void close_plane_info (float *info, float *close_info, int particle_num);

__global__ void initialize_holo_plane (hipfftComplex *holo);
__global__ void particle_volume (float *info, unsigned char *out, float *close_info);
__global__ void trans_func(hipfftComplex *trans, float dist);
__global__ void extract_plane_from_vol(unsigned char* V, hipfftComplex *plane, int num);
__global__ void fftshift_2D(hipfftComplex *data);
__global__ void plane_complex_multiple (hipfftComplex *A, hipfftComplex *B, hipfftComplex *C);
__global__ void holo_to_float_image (hipfftComplex *data, float *image);
__global__ void two_dim_divide_for_fft (hipfftComplex *data);

FILE *fp;
/**********************************main***********************************/
int main(int argc, char** argv){
    printf("%s Starting...\n", argv[0]);
    char *output_dir = argv[1];
    int particle_num = atoi(argv[2]);
    int image_count = atoi(argv[3]);

    fp = fopen(ref_img_path,"rb");
    if(fp == NULL){
        printf("NO REFERENCE IMAGE! quitting...\n");
        exit(1);
    }
    int read_conf;
    read_conf = fread(header_buf, sizeof(unsigned char), 1078, fp);
    fclose(fp);

    float host_diam[particle_num];
    float host_posi_x[particle_num], host_posi_y[particle_num], host_posi_z[particle_num];
    unsigned char *dev_V;
    float *host_particle_info, *dev_particle_info;
    float *dev_float_image, float_image[height/2][width/2];
    unsigned char image_out[height/2][width/2];
    float host_close_info[7], *dev_close_info;
    float dist_to_next_holo;

    int dev = 0;
    hipSetDevice(dev);

    dim3 grid(particle_num, width, height), block(1);
    dim3 grid2(width,height), block2(1);
    dim3 grid3(width/2,height/2), block3(1);

    CHECK(hipMalloc((void **)&dev_particle_info, sizeof(float)*particle_num*4));
    CHECK(hipMalloc((void **)&dev_V, sizeof(unsigned char)*width*height*particle_num));
    CHECK(hipMalloc((void **)&dev_close_info, sizeof(float)*7));
    CHECK(hipMalloc((void **)&dev_float_image, sizeof(float)*width*height/4));

    hipfftHandle plan;
    hipfftPlan2d(&plan,width,height,HIPFFT_C2C);
    hipfftComplex *devc_object, *devc_hologram, *devc_trans;
    CHECK(hipMalloc((void **)&devc_object, sizeof(hipfftComplex)*width*height)); 
    CHECK(hipMalloc((void **)&devc_hologram, sizeof(hipfftComplex)*width*height)); 
    CHECK(hipMalloc((void **)&devc_trans, sizeof(hipfftComplex)*width*height));

    sprintf(output_dir_num,"%s/num_%05d/",output_dir,particle_num);
    printf("%s\n",output_dir_num);
    mkdir(output_dir_num, S_IRWXU | S_IRGRP | S_IXGRP | S_IROTH | S_IXOTH);
    srand((unsigned int)time(NULL));

    for(int count = 0; count < image_count; count++){
        particle_diam_dist(host_diam, particle_num);
        particle_posi_dist(host_posi_x, host_posi_y, host_posi_z, particle_num);
        z_axis_sort(host_posi_z, 0, particle_num-1);

        host_particle_info = (float *)malloc(sizeof(float)*particle_num*4);
        for (int i = 0; i < particle_num; i++){
            host_particle_info[4*i] = host_diam[i];
            host_particle_info[4*i+1] = host_posi_x[i];
            host_particle_info[4*i+2] = host_posi_y[i];
            host_particle_info[4*i+3] = host_posi_z[i];
            printf("%lf %lf %lf %lf\n",host_diam[i],host_posi_x[i],host_posi_y[i],host_posi_z[i]);
        }

        close_plane_info(host_particle_info, host_close_info, particle_num);

        CHECK(hipMemcpy(dev_particle_info, host_particle_info, sizeof(float)*particle_num*4, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(dev_close_info, host_close_info, sizeof(float)*7, hipMemcpyHostToDevice));

        particle_volume<<<grid, block>>>(dev_particle_info, dev_V, dev_close_info);

        extract_plane_from_vol<<<grid2, block2>>>(dev_V, devc_object, 0);
        for (int itr = 0; itr < particle_num - 1; itr++) {
            hipfftExecC2C(plan, devc_object, devc_object, HIPFFT_FORWARD);
            fftshift_2D<<<grid2, block2>>>(devc_object);
            dist_to_next_holo = host_particle_info[4*(itr+1) + 3] - host_particle_info[4*itr + 3];
            trans_func<<<grid2, block2>>>(devc_trans, dist_to_next_holo);
            plane_complex_multiple<<<grid2, block2>>>(devc_object, devc_trans, devc_hologram);
            fftshift_2D<<<grid2, block2>>>(devc_hologram);
            hipfftExecC2C(plan, devc_hologram, devc_hologram, HIPFFT_BACKWARD);
            two_dim_divide_for_fft<<<grid2,block2>>>(devc_hologram);
            extract_plane_from_vol<<<grid2, block2>>>(dev_V, devc_object, itr+1);
            plane_complex_multiple<<<grid2, block2>>>(devc_object, devc_hologram, devc_object);
        }
        hipfftExecC2C(plan, devc_object, devc_object, HIPFFT_FORWARD);
        fftshift_2D<<<grid2, block2>>>(devc_object);
        trans_func<<<grid2, block2>>>(devc_trans,  dist_to_cam + (float)depth*DX - host_particle_info[4*(particle_num-1) + 3]);
        plane_complex_multiple<<<grid2, block2>>>(devc_object, devc_trans, devc_hologram);
        fftshift_2D<<<grid2, block2>>>(devc_hologram);
        hipfftExecC2C(plan, devc_hologram, devc_hologram, HIPFFT_BACKWARD);
        two_dim_divide_for_fft<<<grid2,block2>>>(devc_hologram);

        holo_to_float_image<<<grid3, block3>>>(devc_hologram, dev_float_image);

        CHECK(hipMemcpy(float_image, dev_float_image, sizeof(float)*width*height/4, hipMemcpyDeviceToHost));

        for (int y=0; y < height/2; y++) {
            for (int x=0; x < width/2; x++) {
                image_out[y][x] = (unsigned char)(peak_bright*float_image[y][x]);
            }
        }

        sprintf(output_path, "%s/%05d.bmp",output_dir_num,count);
        fp = fopen(output_path, "wb");
        fwrite(header_buf, sizeof(unsigned char), 1078, fp);
        fwrite(image_out, sizeof(unsigned char), width*height/4, fp);
        fclose(fp);

        printf("%d of %d has processed\n",count+1,image_count);
        printf("\n\n");

    }

    free(host_particle_info);
    hipFree(dev_particle_info);
    hipFree(dev_V);
    hipfftDestroy(plan);
    hipFree(devc_object);
    hipFree(devc_hologram);
    hipFree(devc_trans);
    hipFree(dev_float_image);
    hipFree(dev_close_info);
    hipDeviceReset();

    return 0;
}

void particle_posi_dist (float *x, float *y, float*z, int particle_num){
    for (int i = 0; i < particle_num; i++) {
        x[i] = (rand() % (width/2) + width/4.0) * DX;
        y[i] = (rand() % (height/2) + height/4.0) * DX;
        z[i] = rand() % depth * DX;
    }
}

void particle_diam_dist (float *diam, int index){
    float tmp1, tmp2, tmp3;
    for (int i = 0; i < index ; i++) {
        while(1){
            tmp1 = (float)rand() / (float)RAND_MAX;
            tmp2 = (float)rand() / (float)RAND_MAX;
            tmp3 = sqrt(-2.0*log(tmp1))*cos(2.0*M_PI*tmp2);
            diam[i] = sd_diam*tmp3 + mean_diam;
            if (diam[i] > 0.0) break;
        }
    }
}

void z_axis_sort (float *array, int left, int right){
    int tmp;
    int i = left;
    int j = right;
    float pivot = array[(i+j)/2];
    while(1){
        while(array[i] < pivot)
            i++;
        
        while(array[j] > pivot)
            j--;
        
        if(i >= j)
            break;

        tmp = array[i];
        array[i] = array[j];
        array[j] = tmp;

        i++;
        j--;
    }
        if(i-1>left)
            z_axis_sort(array,left,i-1);

        if(j+1<right)
            z_axis_sort(array,j+1,right);
}

void close_plane_info (float *info, float *close_info, int particle_num){
    close_info[0] = (rand() % (width/4) + 3.0*width/8.0) * DX;
    close_info[1] = (rand() % (height/4) + 3.0*height/8.0) * DX;
    
    float diam[2];
    particle_diam_dist(diam, 2);
    close_info[4] = diam[0];
    close_info[5] = diam[1];

    float phi1, cos1, sin1;
    phi1 = 2.0*PI*(float)(rand() % 1000)/1000.0;
    cos1 = cos(phi1);
    sin1 = sin(phi1);

    float dist_of_two;
    dist_of_two = 2.0*(diam[0] + diam[1])/2.0 + 1.0*(float)(rand() % (int)(diam[0] + diam[1]));

    close_info[2] = close_info[0] + dist_of_two*cos1;
    close_info[3] = close_info[1] + dist_of_two*sin1;

    close_info[6] = (float)(rand() % particle_num);

    printf("x1, y1 : %lf %lf\n",close_info[0],close_info[1]);
    printf("x2, y2 : %lf %lf\n",close_info[2],close_info[3]);
    printf("diam1, diam2 : %lf %lf\n",diam[0],diam[1]);
    printf("dist of two : %lf\n",dist_of_two);
    printf("angle of two [deg] :%lf\n",phi1*180/PI);
    printf("z_idx: %lf \n",close_info[6]);
    printf("z coordinate : %lf\n",info[4*(int)close_info[6]+3]);

    /****
    * 0 : x coordinate of first droplet
    * 1 : y coordinate of first droplet
    * 2 : x coordinate of second droplet
    * 3 : y coordinate of seconda droplet
    * 4 : diameter of first droplet
    * 5 : diameter of second droplet
    * 6 : arbitary selected plane index (indicates corresponding z coordinate given in advance)
    ****/
}

__global__ void particle_volume (float *info, unsigned char *out, float *close_info){
    int x, y, idx;
    x = blockIdx.y;
    y = blockIdx.z;
    idx = blockIdx.x;

    if(idx != (int)close_info[6]){
        if( ((float)x*DX-info[4*idx + 1])*((float)x*DX-info[4*idx + 1]) + ((float)y*DX-info[4*idx + 2])*((float)y*DX-info[4*idx + 2]) > info[4*idx]*info[4*idx]/4.0 ){
            out[x + y*width + idx*width*height] = (unsigned char)1;
        }else{
            out[x + y*width + idx*width*height] = (unsigned char)0;
        }
    }else{
        if( ( ((float)x*DX - close_info[0])*((float)x*DX - close_info[0]) + ((float)y*DX - close_info[1])*((float)y*DX - close_info[1]) < close_info[4]*close_info[4]/4.0 ) || ( ((float)x*DX - close_info[2])*((float)x*DX - close_info[2]) + ((float)y*DX - close_info[3])*((float)y*DX - close_info[3]) < close_info[5]*close_info[5]/4.0 ) ){
            out[x + y*width + (int)close_info[6]*width*height] = (unsigned char)0;
        }else{
            out[x + y*width + (int)close_info[6]*width*height] = (unsigned char)1;
        }
    }
}

__global__  void trans_func(hipfftComplex *trans, float dist){
    int x, y;
    x = blockIdx.x;
    y = blockIdx.y;

    float c0,c1,c2, tmp;

    c0 = 2.0*PI*dist/WAVE_LEN;
    c1 = WAVE_LEN*WAVE_LEN/width/width/DX/DX;
    c2 = WAVE_LEN*WAVE_LEN/height/height/DX/DX;
    
    tmp = c0*sqrt(1.0-c1*((float)x-(float)width/2.0)*((float)x-(float)width/2.0)-c2*((float)y-(float)height/2.0)*((float)y-(float)height/2.0));
    trans[x + y*width].x = cos(tmp);
    trans[x + y*width].y = sin(tmp);
}

__global__ void extract_plane_from_vol(unsigned char* V, hipfftComplex *plane, int num){
    int x,y;
    x = blockIdx.x;
    y = blockIdx.y;

    plane[x + width*y].x = (float)V[x + width*y + width*height*num];
    plane[x + width*y].y = 0.0;
}

__global__ void plane_complex_multiple (hipfftComplex *A, hipfftComplex *B, hipfftComplex *C){
	int x = blockIdx.x;
    int y = blockIdx.y;

    float tmp1, tmp2;

    tmp1 = A[x + width*y].x * B[x + width*y].x - A[x + width*y].y * B[x + width*y].y;
    tmp2 = A[x + width*y].x * B[x + width*y].y + A[x + width*y].y * B[x + width*y].x;

    C[x + width*y].x = tmp1;
    C[x + width*y].y = tmp2;
}

__global__ void fftshift_2D(hipfftComplex *data){
	int x = blockIdx.x;
    int y = blockIdx.y;
    hipfftComplex temp1,temp2;
    
    if((x < width/2) && (y < height/2)){
        temp1 = data[x + width*y];
        data[x + width*y] = data[x + width/2 + width*(y + height/2)];
        data[x + width/2 + width*(y + height/2)] = temp1;
    }
    if((x < width/2) && (y >= height/2)){
        temp2 = data[x + width*y];
        data[x + width*y] = data[x + width/2 + width*(y - height/2)];
        data[x + width/2 + width*(y - height/2)] = temp2;
    }
}

__global__ void initialize_holo_plane (hipfftComplex *holo){
	int x = blockIdx.x;
    int y = blockIdx.y;

    holo[x + width*y].x = 1.0;
    holo[x + width*y].y = 0.0;
}

__global__ void holo_to_float_image (hipfftComplex *data, float *image){
	int x = blockIdx.x;
    int y = blockIdx.y;

    image[x + width/2*y] = sqrt(data[(x + width/4) + width*(y + height/4)].x*data[(x + width/4) + width*(y + height/4)].x + data[(x + width/4) + width*(y + height/4)].y*data[(x + width/4) + width*(y + height/4)].y);

}

__global__ void two_dim_divide_for_fft (hipfftComplex *data){
    int x = blockIdx.x;
    int y = blockIdx.y;

    data[x + width*y].x /= width*height;
    data[x + width*y].y /= width*height;

} 